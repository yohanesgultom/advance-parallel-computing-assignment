// Source adapted from: http://www.programmingsimplified.com/c-program-multiply-matrices

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// create random matrix row-major-format
float* create_flat_matrix(int row, int col, int max)
{
    float* m = (float*)malloc(row*col*sizeof(float));
    int i, j = 0;
    for (i = 0; i < row; i++) {
        for (j = 0; j < col; j++) {
            float val = (max > 0) ? (float)(rand() % max) : 0.0f;
            m[col * i + j] = val;
        }
    }
    return m;
}

// print matrix row-major-format
void print_flat_matrix(float *m, int row, int col)
{
    int i, j = 0;
    for (i = 0; i < row; i++) {
        for (j = 0; j < col; j++) {
            printf("%.2f ", m[col * i + j]);
        }
        printf("\n");
    }
}

void mmul(float *first, int m, int p, float *second, int q, float *multiply)
{
    int c, d, k = 0;
    float sum = .0f;
    for (c = 0; c < m; c++) {
        for (d = 0; d < q; d++) {
            for (k = 0; k < p; k++) {
                sum = sum + first[c*m+k] * second[k*q+d];
            }
            multiply[c*q+d] = sum;
            sum = 0;
        }
    }
}

__global__ void mmul_d(float *first, int m, int p, float *second, int q, float *multiply)
{
    int c, d, k = 0;
    float sum = .0f;
    for (c = 0; c < m; c++) {
        for (d = 0; d < q; d++) {
            for (k = 0; k < p; k++) {
                sum = sum + first[c*m+k] * second[k*q+d];
            }
            multiply[c*q+d] = sum;
            sum = 0;
        }
    }
}

__global__ void mmul_d_thread(float *first, int m, int p, float *second, int q, float *multiply)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int d, k = 0;
    float sum = .0f;
    for (d = 0; d < q; d++) {
        for (k = 0; k < p; k++) {
            sum = sum + first[c*m+k] * second[k*q+d];
        }
        multiply[c*q+d] = sum;
        sum = 0;
    }
}


int main(int argc, char** argv)
{
    if (argc < 6) {
        printf("insufficient args. for A x B = C, required args: [row num A] [col num A OR row num B] [col num B] [cuda block size] [reps]\n");
        return EXIT_FAILURE;
    }

    int m, n, p, q, max = 0;
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    p = n;
    q = atoi(argv[3]);
    int blockSize = atoi(argv[4]);
    int nBlocks = (m * n) / blockSize + ((m * n) % blockSize == 0 ? 0 : 1);
    int reps = atoi(argv[5]);
    max = 10;

    float *first_d, *second_d, *multiply_d;
    float *first, *second, *multiply;

    int i = 0;
    double total_time = 0.0f;
    for (i = 0; i < reps; i++) {
        double exec_time = ((double) clock()) * -1;
        first = create_flat_matrix(m, n, max);
        second = create_flat_matrix(p, q, max);
        multiply = create_flat_matrix(m, q, 0);

        // printf("first:\n");
        // print_flat_matrix(first, m, n);
        //
        // printf("second:\n");
        // print_flat_matrix(second, p, q);

        hipMalloc((void **) &first_d, m * n * sizeof(float));
        hipMalloc((void **) &second_d, p * q * sizeof(float));
        hipMalloc((void **) &multiply_d, m * q * sizeof(float));

        hipMemcpy(first_d, first, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(second_d, second, p * q * sizeof(float), hipMemcpyHostToDevice);

        if (blockSize <= 1) {
            mmul_d <<< 1, 1 >>> (first_d, m, n, second_d, q, multiply_d);
        } else {
            mmul_d_thread <<< nBlocks, blockSize >>> (first_d, m, n, second_d, q, multiply_d);
        }

        hipMemcpy(multiply, multiply_d, m * q * sizeof(float), hipMemcpyDeviceToHost);

        // mmul_h(first, m, n, second, q, multiply);
        // printf("multiply:\n");
        // print_flat_matrix(multiply, m, q);

        free(multiply); free(second); free(first);
        hipFree(first_d); hipFree(second_d); hipFree(multiply_d);
        total_time = total_time + ((exec_time + ((double)clock())) / CLOCKS_PER_SEC);
        // printf("%d: %.6f\n", i, ((exec_time + ((double)clock())) / CLOCKS_PER_SEC));
    }
    printf("%d\t%d\t%d\t%d\t%d\t%.6fs\n", m, n, q, blockSize, reps, (total_time / reps));
    return EXIT_SUCCESS;
}
